#include "hip/hip_runtime.h"
#include "a_mixed_gpu_and_cpu_class.hh"

#include <iostream>

__device__ void a_mixed_gpu_and_cpu_class::a_device_function() {
  float x = 0;
  x += 1;
  printf("%d\n",x);
}


__host__   void a_mixed_gpu_and_cpu_class::a_host_function() {
  float* gpu_mem;
  hipMalloc((void**)&gpu_mem,10*sizeof(float));
  hipFree(gpu_mem);
}


__global__ void some_kernel (double* data) {
  data[0]*=10.0;
}
